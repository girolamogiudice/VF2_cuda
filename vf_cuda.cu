#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <dirent.h>
#include <cstdlib>
#include <string.h>
#include <cutil.h>
#include <shrUtils.h>
#include <cstdio>
#include <windows.h>
#ifdef _WIN32
#include <windows.h>
#elif MACOS
#include <sys/param.h>
#include <sys/sysctl.h>
#else
#include <unistd.h>
#endif



char graph_label[256];
char graph_label_query[256];
int no_of_nodes=0,no_of_nodes_query=0;
int no_of_edge=0,no_of_edge_query=0;FILE *fp;
FILE *fp1;
//Structure to hold a node information

struct edge
	{
		int edge_start;
		int edge_end;
	};
struct __align__(8)  node 
	{
		unsigned long int node_start;
		unsigned long int node_end;
	};
double tot=0;

int *is_already_present;
int *h_result;
int *h_node_label;
int *h_node_label_query;
edge *h_edge;
edge *h_edge_query;
node *h_node;
node *h_node_query;
int maxThreadsPerBlock;
int numcore;
void find(char *path_query,const char *path,char *result_dir);
void read_and_construct_graph(int argc, char** argv);
void read_and_construct_query(int argc, char** argv,const char *path,char *result_dir,float tot);

//void graph_find();
int verify();
int getNumCores();
void create_for_vento(int h_increment,char *path_query,const char *path,char *result_dir);
	
	// d_node,h_node_query[i].node_start,h_node_query[i].node_end,increment,d_result,no_of_edge,d_over);
__global__ void Kernel(node *d_node,int node_start,int node_end,int *increment,int *d_result,int no_of_edge,int *d_present,int maxThreadsPerBlock) 
{
	
	int tid = blockIdx.x*maxThreadsPerBlock + threadIdx.x;
	if(tid<no_of_edge) 
		{
	
			if ( d_node[tid].node_start==node_start && d_node[tid].node_end==node_end || d_node[tid].node_start==node_end && d_node[tid].node_end==node_start) 
				{
					*d_present=1;
					d_result[atomicAdd(increment,1)]=tid;
				}
		}	
}
	

	
	
int main( int argc, char** argv) 
{
	FILE *fpres = fopen("result.txt","w");

	//FILE *fpres = fopen("result.txt","a");
	time_t begin,end;
	char directory[512];
	char result_dir[512];
	begin=clock();
	verify();
	end = clock();
	tot =(double)(end-begin)/CLOCKS_PER_SEC;
	printf("Tempo verifica %.20lf\n",tot);
	int num_of_core=getNumCores();
	int core_to_use;
	printf("\n");
	printf("CPU Analysis");
	printf("\n  Number Of Core:%d\n",num_of_core);
	printf("how many core(s) do you want to use?\n");
	scanf("%d",&core_to_use);
	if(core_to_use>num_of_core) 
		{
			printf("to many cory please insert a number between 1 and %d\n",num_of_core);
			scanf("%d\n",&core_to_use);
		}
	printf("please insert query directory\n");
	scanf("%s",&directory);
	printf("please insert result directory\n");
	scanf("%s",&result_dir);
	read_and_construct_graph( argc, argv);
 	printf("\n----------------------------------------------------------------------------------------------------------------------\n");
	read_and_construct_query(argc,argv,directory,result_dir,tot);
}

void Usage(int argc, char**argv) 
{
	fprintf(stderr,"Usage: %s <graph> \n", argv[0]);
}

int verify()
{
	hipDevice_t dev;
	int major = 0, minor = 0;
	int deviceCount = 0;
	char deviceName[256];

	// note your project will need to link with cuda.lib files on windows
	printf("CUDA Device Query (Driver API) statically linked version \n");
		hipError_t err = hipInit(0);

    CU_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));
	if (deviceCount == 0) 
	{
		printf("There is no device supporting CUDA\n");		
	}
    for (dev = 0; dev < deviceCount; ++dev) 
		{
			CU_SAFE_CALL_NO_SYNC( hipDeviceComputeCapability(&major, &minor, dev) );

				if (dev == 0) 
					{
					// This function call returns 9999 for both major & minor fields, if no CUDA capable devices are present
						if (major == 9999 && minor == 9999)
							printf("There is no device supporting CUDA.\n");
						else if (deviceCount == 1)
							printf("There is 1 device supporting CUDA\n");
						else
							printf("There are %d devices supporting CUDA\n", deviceCount);
					}
			CU_SAFE_CALL_NO_SYNC( hipDeviceGetName(deviceName, 256, dev) );
			printf("\nDevice %d: \"%s\"\n", dev, deviceName);
			printf("  CUDA Capability Major/Minor version number:    %d.%d\n", major, minor);
			if(major==1 && minor==0)
				printf("sorry but graph match requires Cuda Capabilities >=1.1\n");
			int multiProcessorCount;
			CU_SAFE_CALL_NO_SYNC( hipDeviceGetAttribute( &multiProcessorCount, hipDeviceAttributeMultiprocessorCount, dev ) );
			printf("  Multiprocessors x Cores/MP = Cores:            %d (MP) x %d (Cores/MP) = %d (Cores)\n", 
					 multiProcessorCount, ConvertSMVer2Cores(major, minor), 
					 ConvertSMVer2Cores(major, minor) * multiProcessorCount);
			int totalConstantMemory;
			CU_SAFE_CALL_NO_SYNC( hipDeviceGetAttribute( &totalConstantMemory, hipDeviceAttributeTotalConstantMemory, dev ) );
			printf("  Total amount of constant memory:               %u bytes\n", totalConstantMemory);
			int sharedMemPerBlock;
			CU_SAFE_CALL_NO_SYNC( hipDeviceGetAttribute( &sharedMemPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, dev ) );
			printf("  Total amount of shared memory per block:       %u bytes\n", sharedMemPerBlock);
			int regsPerBlock;
			CU_SAFE_CALL_NO_SYNC( hipDeviceGetAttribute( &regsPerBlock, hipDeviceAttributeMaxRegistersPerBlock, dev ) );
			printf("  Total number of registers available per block: %d\n", regsPerBlock);
			int warpSize;
			CU_SAFE_CALL_NO_SYNC( hipDeviceGetAttribute( &warpSize, hipDeviceAttributeWarpSize, dev ) );
			printf("  Warp size:                                     %d\n",	warpSize);
			
			CU_SAFE_CALL_NO_SYNC( hipDeviceGetAttribute( &maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, dev ) );
			printf("  Maximum number of threads per block:           %d\n",	maxThreadsPerBlock);
			int blockDim[3];
			CU_SAFE_CALL_NO_SYNC( hipDeviceGetAttribute( &blockDim[0], hipDeviceAttributeMaxBlockDimX, dev ) );
			CU_SAFE_CALL_NO_SYNC( hipDeviceGetAttribute( &blockDim[1], hipDeviceAttributeMaxBlockDimY, dev ) );
			CU_SAFE_CALL_NO_SYNC( hipDeviceGetAttribute( &blockDim[2], hipDeviceAttributeMaxBlockDimZ, dev ) );
			printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n", blockDim[0], blockDim[1], blockDim[2]);
			int gridDim[3];
			CU_SAFE_CALL_NO_SYNC( hipDeviceGetAttribute( &gridDim[0], hipDeviceAttributeMaxGridDimX, dev ) );
			CU_SAFE_CALL_NO_SYNC( hipDeviceGetAttribute( &gridDim[1], hipDeviceAttributeMaxGridDimY, dev ) );
			CU_SAFE_CALL_NO_SYNC( hipDeviceGetAttribute( &gridDim[2], hipDeviceAttributeMaxGridDimZ, dev ) );
			printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n", gridDim[0], gridDim[1], gridDim[2]);
			int clockRate;
			CU_SAFE_CALL_NO_SYNC( hipDeviceGetAttribute( &clockRate, hipDeviceAttributeClockRate, dev ) );
			printf("  Clock rate:                                    %.2f GHz\n", clockRate * 1e-6f);
}
return(0);
}

void read_and_construct_graph( int argc, char** argv) 
{
    char *input_f;
	if(argc!=2) 
	{
		Usage(argc, argv);
		exit(0);
	}
	input_f = argv[1];
	printf("\n");
	printf("**********Reading Graph File**********\n");
	//Read in Graph from a file
	fp = fopen(input_f,"r");
	if(!fp)
		{
			printf("Error Reading graph file\n");
			return;
		}	
	fscanf(fp,"%s",graph_label);
	fscanf(fp,"%d",&no_of_nodes);
	h_node_label = (int*) malloc(sizeof(int)*no_of_nodes);
	int label=0;
	
	//reading and construct graph
	for( int i = 0; i < no_of_nodes; i++) 
		{	
			fscanf(fp,"%d",&label);
			h_node_label[i] = label;
		}
	fscanf(fp,"%d",&no_of_edge);
	h_edge = (edge*) malloc(sizeof(edge)*no_of_edge);
	int start;
	int end;
	//printf("no of edge %d\n",no_of_edge);
	for(int i=0;i<no_of_edge;i++) 
	{
		fscanf(fp,"%d %d",&start,&end);
		h_edge[i].edge_start = start ;
		h_edge[i].edge_end = end ;
	//printf("result %d	%d	\n",h_edge[i].edge_start,h_edge[i].edge_end);

	}
	fclose(fp);
	printf("graph name: %s \nno. of node: %d\nno. of edge: %d\n",graph_label,no_of_nodes,no_of_edge); 

}
void read_and_construct_query(int argc, char** argv,const char *path,char *result_dir,float tot)
{	
	
	char *input_f1;
	DIR *pdir = NULL; // remember, it's good practice to initialise a pointer to NULL!
    pdir = opendir (path); // "." will refer to the current directory
    struct dirent *pent = NULL;
    if (pdir == NULL) // if pdir wasn't initialised correctly
		{ // print an error message and exit the program
			printf ("\nERROR! pdir could not be initialised correctly");
			return; // exit the function
		} // end if
 
    while (pent = readdir (pdir)) // while there is still something in the directory to list
		{	
			int start_query=0;
			int end_query=0;
			int label_query=0;
			char str[128];
			strcpy (str,path);
				if (pent == NULL) // if pent has not been initialised correctly
					{ // print an error message, and exit the program
						printf ("\nERROR! pent could not be initialised correctly");
						return; // exit the function
					}
					printf("query %s\n",pent->d_name);
				if (strstr(pent->d_name,"txt")) 
					{

						strcat (str,pent->d_name);
						input_f1 = str;
						printf("**********Reading Query File**********\n");
						//Read in Graph from a file
						fp1 = fopen(input_f1,"r");
						if (!fp1)
						{
							printf("Error Reading graph file\n");
							return;
						} 
		

			//reading and construct graph query
			fscanf(fp1,"%s",graph_label_query);
			fscanf(fp1,"%d",&no_of_nodes_query);
			h_node_label_query = (int*) malloc(sizeof(int)*no_of_nodes_query);
			for ( int i = 0; i < no_of_nodes_query; i++) 
				{
					fscanf(fp1,"%d",&label_query);
					h_node_label_query[i] = label_query;
					//printf("h_node_query %d\n",h_node_label_query[i]);
				}

			fscanf(fp1,"%d",&no_of_edge_query);
			h_edge_query = (edge*) malloc(sizeof(edge)*no_of_edge_query);

			//printf("no of edge %d\n",no_of_edge_query);
			for	(int i=0;i<no_of_edge_query;i++) 
				{
					fscanf(fp1,"%d %d",&start_query,&end_query);
					h_edge_query[i].edge_start= start_query ;
					h_edge_query[i].edge_end = end_query ;	
					//printf("h_edge start %d , h_edge end %d \n",h_edge_query[i].edge_start,h_edge_query[i].edge_end);
				}

			fclose(fp1);
			printf("**********graph & query struct constructed**********\n");

			is_already_present=(int*) malloc(sizeof(int)*no_of_edge_query);
			node query;
			for(int i=0;i<no_of_edge_query;i++)
				is_already_present[i]=0;

			h_node = (node*) malloc(sizeof(node)*no_of_edge);
			h_node_query = (node*) malloc(sizeof(node)*no_of_edge_query);

			for (int i=0;i<no_of_edge;i++) 
				{
					h_node[i].node_start=h_node_label[h_edge[i].edge_start];
					h_node[i].node_end=h_node_label[h_edge[i].edge_end];
					//printf("h_node start %d ,h_node end %d\n",h_node[i].node_start,h_node[i].node_end);
				}

			for (int i=0;i<no_of_edge_query;i++) 
				{
					h_node_query[i].node_start=h_node_label_query[h_edge_query[i].edge_start];
					h_node_query[i].node_end=h_node_label_query[h_edge_query[i].edge_end];
					//printf("h_node start %d ,h_node end %d\n",h_node_query[i].node_start,h_node_query[i].node_end);
				}				

			for ( int i=0;i<no_of_edge_query;i++) 
				{
					query.node_start=h_node_query[i].node_start;
					query.node_end=h_node_query[i].node_end;
					//printf("node start %d node end %d\n",query.node_start,query.node_end);
				for (int j=0;j<no_of_edge_query;j++) 
					{
						if (j==i) 
							continue ;
						if ( query.node_start==h_node_query[j].node_start && query.node_end==h_node_query[j].node_end || query.node_start==h_node_query[j].node_end && query.node_end==h_node_query[j].node_start )
							is_already_present[j]=1;
					}				
					is_already_present[i]=0;
				}
			find(str,path,result_dir);
					}
		}
  closedir (pdir);
 
}

void find(char *path_query,const char *path,char *result_dir) 
{	
	
	
	int is_not_present=0;
	int h_increment=0;
	int num_of_blocks = 1;
	int num_of_threads_per_block = maxThreadsPerBlock;
	int *d_result;
	int h_present;
	int *d_present;
	int *d_increment;
	node *d_node;
	node *d_node_query;
	time_t begin,end;
	h_result = (int*) malloc(sizeof(int)*5000); //max number of occurrency in a graph
				
		if(no_of_edge>maxThreadsPerBlock)
			{
				num_of_blocks = (int)ceil(no_of_edge/(double)maxThreadsPerBlock); 
				num_of_threads_per_block = maxThreadsPerBlock; 
			}
	//Copy the Node list to device memory

		
	//print information about graph target and query
	printf("\nquery name: %s\nno. of node query: %d\nno. of edge query %d\n",graph_label_query,no_of_nodes_query,no_of_edge_query); 


	begin = clock();	
	
	hipMalloc( (void**) &d_increment, sizeof(int));
	hipMalloc( (void**) &d_present, sizeof(int));
	hipMalloc( (void**) &d_node, sizeof(node)*no_of_edge) ;
	hipMemcpy( d_node, h_node, sizeof(node)*no_of_edge, hipMemcpyHostToDevice) ;
	hipMalloc( (void**) &d_node_query, sizeof(node)*no_of_edge_query) ;
	hipMemcpy( d_node_query, h_node_query, sizeof(node)*no_of_edge_query, hipMemcpyHostToDevice) ;
	hipMalloc( (void**) &d_result, sizeof(int)*5000) ; //max number of occurrency in a graph (see line 288)
	hipMemcpy( d_result, h_result, sizeof(int)*5000, hipMemcpyHostToDevice) ; //max number of occurrency in a graph(see line 288)
	hipMemcpy( d_increment, &h_increment, sizeof(int), hipMemcpyHostToDevice) ;



	for(int i=0;i<no_of_edge_query;i++) 
		{
			if (is_already_present[i]==1) 
				continue;
			h_present=0 ;
			//printf("node start %d node end %d\n",h_node_query[i].node_start,h_node_query[i].node_end);
			//printf("i %d ,h_present %d,node start %d node end %d\n",i,h_present,h_node_query[i].node_start,h_node_query[i].node_end,increment);
			hipMemcpy( d_present, &h_present, sizeof(int), hipMemcpyHostToDevice) ;
			Kernel<<<num_of_blocks,num_of_threads_per_block >>>( d_node,h_node_query[i].node_start,h_node_query[i].node_end,d_increment,d_result,no_of_edge,d_present,maxThreadsPerBlock);
			hipDeviceSynchronize();
			hipMemcpy( &h_present, d_present, sizeof(int), hipMemcpyDeviceToHost) ;
		

			if (h_present== 0 ) 
				{
					printf("query non presente\n");
					printf("----------------------------------------------------------------------------------------------------------------------\n");
					
					is_not_present=1;
					break ;
					//exit(0);

				}
		}	
	
	
	hipMemcpy( h_result, d_result, sizeof(int)*5000, hipMemcpyDeviceToHost) ;
	
	
	end = clock();
	hipMemcpy( &h_increment, d_increment, sizeof(int), hipMemcpyDeviceToHost) ;
	
	hipFree(d_node);
	hipFree(d_increment);
	hipFree(d_result);
	hipFree(d_present);
	hipFree(d_node_query);
	free(h_node);
	
		//printf("increment dopo %d\n",h_increment);
	float time_subgraph = (double)(end-begin)/CLOCKS_PER_SEC;
	printf("subgraph created in: %.20lf\n",time_subgraph);

	if (is_not_present!=1) 
		{	
			create_for_vento( h_increment,path_query,path,result_dir);
		}
}


void create_for_vento(int h_increment,char *path_query,const char *path,char *result_dir) 
{
	edge *h_vento_edge = (edge*) malloc(sizeof(edge)*h_increment);
	int *nodi=(int *)malloc(sizeof(int)*10000);
	time_t begin,end;
	int edge;
	int *result=(int *)malloc(sizeof(int)*2*h_increment);
	for (int i=0;i<h_increment;i++) 
		result[i]=h_edge[h_result[i]].edge_start;
	for (int i=h_increment;i<h_increment*2;i++) 
		result[i]=h_edge[h_result[i-h_increment]].edge_end;
	//ho unito i 2 indici dimensione 2*h_increment

	int *change=(int *)malloc(2*h_increment*sizeof(int));
	for (int i=0;i<2*h_increment;i++) 
		{
			change[i]=0;
		}

	int data=0;
	int index=0;
	for(int i=0;i<2*h_increment;i++) 
		{
			if (change[i]==1)  
				continue ;

			edge=result[i];
			nodi[index]=edge;
			index++;


			for (int j=0;j<2*h_increment;j++) 
				{
			
					if( edge == result[j] && change[j]==0) 
						{
							result[j]=data;
							change[j]=1;
						}

				}
			data++;

		}
		//ho creato result[i] che è un vettore che si deve adattare a sing che diviso per 2 mi darà arco iniziale e finale.


	for (int j=0;j<h_increment;j++) 
		h_vento_edge[j].edge_start=result[j];
	for (int i=0;i<h_increment;i++)
		h_vento_edge[i].edge_end=result[i+h_increment];


	char query[]="#";
	char name_file[512];
	strcpy (name_file,path);

			


	FILE *fpo = fopen("graph.txt","w");
	fprintf(fpo,"%s \n",query);
	fprintf(fpo,"%d \n",data);
	for(int j=0;j<data;j++)
		fprintf(fpo,"%d\n",h_node_label[nodi[j]]);
	fprintf(fpo,"%d \n",h_increment);
	for(int i=0;i<h_increment;i++)
		fprintf(fpo,"%d %d\n",h_vento_edge[i].edge_start,h_vento_edge[i].edge_end);	
	fclose(fpo);
	printf("subgraph composed of %d node and %d edge\n",data,h_increment);

	char strvento[80];

	strcpy (strvento,"Vento.exe");
	strcat (strvento," graph.txt ");
 
	strcat (strvento," ");
	strcat (strvento,path_query);
	begin=clock();
	###########here call vento. It works only for windows##########
	system(strvento);
	end=clock();
	float time_vento = (double)(end-begin)/CLOCKS_PER_SEC;
	printf("vento match in: %.20lf\n",time_vento);
	
 	printf("----------------------------------------------------------------------------------------------------------------------\n");
	
	

}

int getNumCores() 
{
	#ifdef WIN32
    SYSTEM_INFO sysinfo;
    GetSystemInfo(&sysinfo);
    return sysinfo.dwNumberOfProcessors;
	#elif MACOS
		int nm[2];
		size_t len = 4;
		uint32_t count;

		nm[0] = CTL_HW; nm[1] = HW_AVAILCPU;
		sysctl(nm, 2, &count, &len, NULL, 0);

    if(count < 1) 	
		{
			nm[1] = HW_NCPU;
			sysctl(nm, 2, &count, &len, NULL, 0);
			if(count < 1) { count = 1; }
		}
    return count;
	#else
		return sysconf(_SC_NPROCESSORS_ONLN);
	#endif
}
